#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2020 by Contributors
 * \file array/cuda/coo_sort.cc
 * \brief Sort COO index
 */
#include <dgl/array.h>
#include "../../runtime/cuda/cuda_common.h"
#include "./utils.h"

namespace dgl {

using runtime::NDArray;

namespace aten {
namespace impl {

///////////////////////////// COOSort_ /////////////////////////////

template <DLDeviceType XPU, typename IdType>
void COOSort_(COOMatrix* coo, bool sort_column) {
  // TODO(minjie): Current implementation is based on cusparse which only supports
  //   int32_t. To support int64_t, we could use the Radix sort algorithm provided
  //   by CUB.
  CHECK(sizeof(IdType) == 4) << "CUDA COOSort does not support int64.";
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  auto device = runtime::DeviceAPI::Get(coo->row->ctx);
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }
  CUSPARSE_CALL(hipsparseSetStream(thr_entry->cusparse_handle, thr_entry->stream));


  NDArray row = coo->row;
  NDArray col = coo->col;
  if (!aten::COOHasData(*coo))
    coo->data = aten::Range(0, row->shape[0], row->dtype.bits, row->ctx);
  NDArray data = coo->data;
  int32_t* row_ptr = static_cast<int32_t*>(row->data);
  int32_t* col_ptr = static_cast<int32_t*>(col->data);
  int32_t* data_ptr = static_cast<int32_t*>(data->data);

  // sort row
  size_t workspace_size = 0;
  CUSPARSE_CALL(hipsparseXcoosort_bufferSizeExt(
      thr_entry->cusparse_handle,
      coo->num_rows, coo->num_cols,
      row->shape[0],
      row_ptr,
      col_ptr,
      &workspace_size));
  void* workspace = device->AllocWorkspace(row->ctx, workspace_size);
  CUSPARSE_CALL(hipsparseXcoosortByRow(
      thr_entry->cusparse_handle,
      coo->num_rows, coo->num_cols,
      row->shape[0],
      row_ptr,
      col_ptr,
      data_ptr,
      workspace));
  device->FreeWorkspace(row->ctx, workspace);

  if (sort_column) {
    // First create a row indptr array and then call csrsort
    int32_t* indptr = static_cast<int32_t*>(
        device->AllocWorkspace(row->ctx, (coo->num_rows + 1) * sizeof(IdType)));
    CUSPARSE_CALL(hipsparseXcoo2csr(
          thr_entry->cusparse_handle,
          row_ptr,
          row->shape[0],
          coo->num_rows,
          indptr,
          HIPSPARSE_INDEX_BASE_ZERO));
    CUSPARSE_CALL(hipsparseXcsrsort_bufferSizeExt(
          thr_entry->cusparse_handle,
          coo->num_rows,
          coo->num_cols,
          row->shape[0],
          indptr,
          col_ptr,
          &workspace_size));
    void* workspace = device->AllocWorkspace(row->ctx, workspace_size);
    hipsparseMatDescr_t descr;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
    CUSPARSE_CALL(hipsparseXcsrsort(
          thr_entry->cusparse_handle,
          coo->num_rows,
          coo->num_cols,
          row->shape[0],
          descr,
          indptr,
          col_ptr,
          data_ptr,
          workspace));
    CUSPARSE_CALL(hipsparseDestroyMatDescr(descr));
    device->FreeWorkspace(row->ctx, workspace);
    device->FreeWorkspace(row->ctx, indptr);
  }

  coo->row_sorted = true;
  coo->col_sorted = sort_column;
}

template void COOSort_<kDLGPU, int32_t>(COOMatrix* coo, bool sort_column);
template void COOSort_<kDLGPU, int64_t>(COOMatrix* coo, bool sort_column);

///////////////////////////// COOIsSorted /////////////////////////////

template <typename IdType>
__global__ void _COOIsSortedKernel(
    const IdType* row, const IdType* col,
    int64_t nnz, int8_t* row_sorted, int8_t* col_sorted) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;
  while (tx < nnz) {
    if (tx == 0) {
      row_sorted[0] = 1;
      col_sorted[0] = 1;
    } else {
      row_sorted[tx] = static_cast<int8_t>(row[tx - 1] <= row[tx]);
      col_sorted[tx] = static_cast<int8_t>(
          row[tx - 1] < row[tx] || col[tx - 1] <= col[tx]);
    }
    tx += stride_x;
  }
}

template <DLDeviceType XPU, typename IdType>
std::pair<bool, bool> COOIsSorted(COOMatrix coo) {
  const int64_t nnz = coo.row->shape[0];
  const auto& ctx = coo.row->ctx;
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  auto device = runtime::DeviceAPI::Get(ctx);
  // We allocate a workspace of 2*nnz bytes. It wastes a little bit memory but should
  // be fine.
  int8_t* row_flags = static_cast<int8_t*>(device->AllocWorkspace(ctx, nnz));
  int8_t* col_flags = static_cast<int8_t*>(device->AllocWorkspace(ctx, nnz));
  const int nt = cuda::FindNumThreads(nnz);
  const int nb = (nnz + nt - 1) / nt;
  _COOIsSortedKernel<<<nb, nt, 0, thr_entry->stream>>>(
      coo.row.Ptr<IdType>(), coo.col.Ptr<IdType>(),
      nnz, row_flags, col_flags);

  const bool row_sorted = cuda::AllTrue(row_flags, nnz, ctx);
  const bool col_sorted = row_sorted? cuda::AllTrue(col_flags, nnz, ctx) : false;

  device->FreeWorkspace(ctx, row_flags);
  device->FreeWorkspace(ctx, col_flags);

  return {row_sorted, col_sorted};
}

template std::pair<bool, bool> COOIsSorted<kDLGPU, int32_t>(COOMatrix coo);
template std::pair<bool, bool> COOIsSorted<kDLGPU, int64_t>(COOMatrix coo);

}  // namespace impl
}  // namespace aten
}  // namespace dgl
