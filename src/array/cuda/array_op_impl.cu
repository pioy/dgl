#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2020 by Contributors
 * \file array/cuda/array_op_impl.cu
 * \brief Array operator GPU implementation
 */
#include <dgl/array.h>
#include "../../runtime/cuda/cuda_common.h"
#include "./utils.h"
#include "../arith.h"

namespace dgl {
using runtime::NDArray;
namespace aten {
namespace impl {

///////////////////////////// BinaryElewise /////////////////////////////

template <typename IdType, typename Op>
__global__ void _BinaryElewiseKernel(
    const IdType* lhs, const IdType* rhs, IdType* out, int64_t length) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[tx] = Op::Call(lhs[tx], rhs[tx]);
    tx += stride_x;
  }
}

template <DLDeviceType XPU, typename IdType, typename Op>
IdArray BinaryElewise(IdArray lhs, IdArray rhs) {
  const int64_t len = lhs->shape[0];
  IdArray ret = NewIdArray(lhs->shape[0], lhs->ctx, lhs->dtype.bits);
  const IdType* lhs_data = static_cast<IdType*>(lhs->data);
  const IdType* rhs_data = static_cast<IdType*>(rhs->data);
  IdType* ret_data = static_cast<IdType*>(ret->data);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  int nt = cuda::FindNumThreads(len);
  int nb = (len + nt - 1) / nt;
  _BinaryElewiseKernel<IdType, Op><<<nb, nt, 0, thr_entry->stream>>>(
      lhs_data, rhs_data, ret_data, len);
  return ret;
}

template IdArray BinaryElewise<kDLGPU, int32_t, arith::Add>(IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::Sub>(IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::Mul>(IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::Div>(IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::GT>(IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::LT>(IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::GE>(IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::LE>(IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::EQ>(IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::NE>(IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::Add>(IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::Sub>(IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::Mul>(IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::Div>(IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::GT>(IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::LT>(IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::GE>(IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::LE>(IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::EQ>(IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::NE>(IdArray lhs, IdArray rhs);


template <typename IdType, typename Op>
__global__ void _BinaryElewiseKernel(
    const IdType* lhs, IdType rhs, IdType* out, int64_t length) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[tx] = Op::Call(lhs[tx], rhs);
    tx += stride_x;
  }
}

template <DLDeviceType XPU, typename IdType, typename Op>
IdArray BinaryElewise(IdArray lhs, IdType rhs) {
  const int64_t len = lhs->shape[0];
  IdArray ret = NewIdArray(lhs->shape[0], lhs->ctx, lhs->dtype.bits);
  const IdType* lhs_data = static_cast<IdType*>(lhs->data);
  IdType* ret_data = static_cast<IdType*>(ret->data);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  int nt = cuda::FindNumThreads(len);
  int nb = (len + nt - 1) / nt;
  _BinaryElewiseKernel<IdType, Op><<<nb, nt, 0, thr_entry->stream>>>(
      lhs_data, rhs, ret_data, len);
  return ret;
}

template IdArray BinaryElewise<kDLGPU, int32_t, arith::Add>(IdArray lhs, int32_t rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::Sub>(IdArray lhs, int32_t rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::Mul>(IdArray lhs, int32_t rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::Div>(IdArray lhs, int32_t rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::GT>(IdArray lhs, int32_t rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::LT>(IdArray lhs, int32_t rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::GE>(IdArray lhs, int32_t rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::LE>(IdArray lhs, int32_t rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::EQ>(IdArray lhs, int32_t rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::NE>(IdArray lhs, int32_t rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::Add>(IdArray lhs, int64_t rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::Sub>(IdArray lhs, int64_t rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::Mul>(IdArray lhs, int64_t rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::Div>(IdArray lhs, int64_t rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::GT>(IdArray lhs, int64_t rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::LT>(IdArray lhs, int64_t rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::GE>(IdArray lhs, int64_t rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::LE>(IdArray lhs, int64_t rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::EQ>(IdArray lhs, int64_t rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::NE>(IdArray lhs, int64_t rhs);



template <typename IdType, typename Op>
__global__ void _BinaryElewiseKernel(
    IdType lhs, const IdType* rhs, IdType* out, int64_t length) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[tx] = Op::Call(lhs, rhs[tx]);
    tx += stride_x;
  }
}

template <DLDeviceType XPU, typename IdType, typename Op>
IdArray BinaryElewise(IdType lhs, IdArray rhs) {
  const int64_t len = rhs->shape[0];
  IdArray ret = NewIdArray(rhs->shape[0], rhs->ctx, rhs->dtype.bits);
  const IdType* rhs_data = static_cast<IdType*>(rhs->data);
  IdType* ret_data = static_cast<IdType*>(ret->data);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  int nt = cuda::FindNumThreads(len);
  int nb = (len + nt - 1) / nt;
  _BinaryElewiseKernel<IdType, Op><<<nb, nt, 0, thr_entry->stream>>>(
      lhs, rhs_data, ret_data, len);
  return ret;
}

template IdArray BinaryElewise<kDLGPU, int32_t, arith::Add>(int32_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::Sub>(int32_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::Mul>(int32_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::Div>(int32_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::GT>(int32_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::LT>(int32_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::GE>(int32_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::LE>(int32_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::EQ>(int32_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int32_t, arith::NE>(int32_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::Add>(int64_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::Sub>(int64_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::Mul>(int64_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::Div>(int64_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::GT>(int64_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::LT>(int64_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::GE>(int64_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::LE>(int64_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::EQ>(int64_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDLGPU, int64_t, arith::NE>(int64_t lhs, IdArray rhs);

template <typename IdType, typename Op>
__global__ void _UnaryElewiseKernel(
    const IdType* lhs, IdType* out, int64_t length) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[tx] = Op::Call(lhs[tx]);
    tx += stride_x;
  }
}

template <DLDeviceType XPU, typename IdType, typename Op>
IdArray UnaryElewise(IdArray lhs) {
  const int64_t len = lhs->shape[0];
  IdArray ret = NewIdArray(lhs->shape[0], lhs->ctx, lhs->dtype.bits);
  const IdType* lhs_data = static_cast<IdType*>(lhs->data);
  IdType* ret_data = static_cast<IdType*>(ret->data);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  int nt = cuda::FindNumThreads(len);
  int nb = (len + nt - 1) / nt;
  _UnaryElewiseKernel<IdType, Op><<<nb, nt, 0, thr_entry->stream>>>(
      lhs_data, ret_data, len);
  return ret;
}

template IdArray UnaryElewise<kDLGPU, int32_t, arith::Neg>(IdArray lhs);
template IdArray UnaryElewise<kDLGPU, int64_t, arith::Neg>(IdArray lhs);

///////////////////////////// Full /////////////////////////////

template <typename IdType>
__global__ void _FullKernel(
    IdType* out, int64_t length, IdType val) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[tx] = val;
    tx += stride_x;
  }
}

template <DLDeviceType XPU, typename IdType>
IdArray Full(IdType val, int64_t length, DLContext ctx) {
  IdArray ret = NewIdArray(length, ctx, sizeof(IdType) * 8);
  IdType* ret_data = static_cast<IdType*>(ret->data);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  int nt = cuda::FindNumThreads(length);
  int nb = (length + nt - 1) / nt;
  _FullKernel<IdType><<<nb, nt, 0, thr_entry->stream>>>(ret_data, length, val);
  return ret;
}

template IdArray Full<kDLGPU, int32_t>(int32_t val, int64_t length, DLContext ctx);
template IdArray Full<kDLGPU, int64_t>(int64_t val, int64_t length, DLContext ctx);


///////////////////////////// Range /////////////////////////////

template <typename IdType>
__global__ void _RangeKernel(IdType* out, IdType low, IdType length) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[tx] = low + tx;
    tx += stride_x;
  }
}

template <DLDeviceType XPU, typename IdType>
IdArray Range(IdType low, IdType high, DLContext ctx) {
  CHECK(high >= low) << "high must be bigger than low";
  const IdType length = high - low;
  IdArray ret = NewIdArray(length, ctx, sizeof(IdType) * 8);
  if (length == 0)
    return ret;
  IdType* ret_data = static_cast<IdType*>(ret->data);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  int nt = cuda::FindNumThreads(length);
  int nb = (length + nt - 1) / nt;
  _RangeKernel<IdType><<<nb, nt, 0, thr_entry->stream>>>(ret_data, low, length);
  return ret;
}

template IdArray Range<kDLGPU, int32_t>(int32_t, int32_t, DLContext);
template IdArray Range<kDLGPU, int64_t>(int64_t, int64_t, DLContext);

///////////////////////////// AsNumBits /////////////////////////////

template <typename InType, typename OutType>
__global__ void _CastKernel(const InType* in, OutType* out, size_t length) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[tx] = in[tx];
    tx += stride_x;
  }
}

template <DLDeviceType XPU, typename IdType>
IdArray AsNumBits(IdArray arr, uint8_t bits) {
  const std::vector<int64_t> shape(arr->shape, arr->shape + arr->ndim);
  IdArray ret = IdArray::Empty(shape, DLDataType{kDLInt, bits, 1}, arr->ctx);
  const int64_t length = ret.NumElements();
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  int nt = cuda::FindNumThreads(length);
  int nb = (length + nt - 1) / nt;
  if (bits == 32) {
    _CastKernel<IdType, int32_t><<<nb, nt, 0, thr_entry->stream>>>(
        static_cast<IdType*>(arr->data), static_cast<int32_t*>(ret->data), length);
  } else {
    _CastKernel<IdType, int64_t><<<nb, nt, 0, thr_entry->stream>>>(
        static_cast<IdType*>(arr->data), static_cast<int64_t*>(ret->data), length);
  }
  return ret;
}


template IdArray AsNumBits<kDLGPU, int32_t>(IdArray arr, uint8_t bits);
template IdArray AsNumBits<kDLGPU, int64_t>(IdArray arr, uint8_t bits);

}  // namespace impl
}  // namespace aten
}  // namespace dgl
