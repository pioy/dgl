#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2019 by Contributors
 * \file array/cpu/array_index_select.cu
 * \brief Array index select GPU implementation
 */
#include <dgl/array.h>
#include "../../runtime/cuda/cuda_common.h"
#include "./utils.h"

namespace dgl {
using runtime::NDArray;
namespace aten {
namespace impl {

template <typename DType, typename IdType>
__global__ void _IndexSelectKernel(const DType* array, const IdType* index,
                                   int64_t length, DType* out) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[tx] = array[index[tx]];
    tx += stride_x;
  }
}

template<DLDeviceType XPU, typename DType, typename IdType>
NDArray IndexSelect(NDArray array, IdArray index) {
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  const DType* array_data = static_cast<DType*>(array->data);
  const IdType* idx_data = static_cast<IdType*>(index->data);
  const int64_t arr_len = array->shape[0];
  const int64_t len = index->shape[0];
  NDArray ret = NDArray::Empty({len}, array->dtype, array->ctx);
  if (len == 0)
    return ret;
  DType* ret_data = static_cast<DType*>(ret->data);
  const int nt = cuda::FindNumThreads(len);
  const int nb = (len + nt - 1) / nt;
  _IndexSelectKernel<<<nb, nt, 0, thr_entry->stream>>>(array_data, idx_data, len, ret_data);
  return ret;
}

template NDArray IndexSelect<kDLGPU, int32_t, int32_t>(NDArray, IdArray);
template NDArray IndexSelect<kDLGPU, int32_t, int64_t>(NDArray, IdArray);
template NDArray IndexSelect<kDLGPU, int64_t, int32_t>(NDArray, IdArray);
template NDArray IndexSelect<kDLGPU, int64_t, int64_t>(NDArray, IdArray);
template NDArray IndexSelect<kDLGPU, float, int32_t>(NDArray, IdArray);
template NDArray IndexSelect<kDLGPU, float, int64_t>(NDArray, IdArray);
template NDArray IndexSelect<kDLGPU, double, int32_t>(NDArray, IdArray);
template NDArray IndexSelect<kDLGPU, double, int64_t>(NDArray, IdArray);

template <DLDeviceType XPU, typename DType>
DType IndexSelect(NDArray array, int64_t index) {
  auto device = runtime::DeviceAPI::Get(array->ctx);
  DType ret = 0;
  device->CopyDataFromTo(
      static_cast<DType*>(array->data) + index, 0, &ret, 0,
      sizeof(DType), array->ctx, DLContext{kDLCPU, 0},
      array->dtype, nullptr);
  return ret;
}

template int32_t IndexSelect<kDLGPU, int32_t>(NDArray array, int64_t index);
template int64_t IndexSelect<kDLGPU, int64_t>(NDArray array, int64_t index);
template uint32_t IndexSelect<kDLGPU, uint32_t>(NDArray array, int64_t index);
template uint64_t IndexSelect<kDLGPU, uint64_t>(NDArray array, int64_t index);
template float IndexSelect<kDLGPU, float>(NDArray array, int64_t index);
template double IndexSelect<kDLGPU, double>(NDArray array, int64_t index);

}  // namespace impl
}  // namespace aten
}  // namespace dgl
