#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2020 by Contributors
 * \file array/cuda/csr_sort.cc
 * \brief Sort COO index
 */
#include <dgl/array.h>
#include "../../runtime/cuda/cuda_common.h"
#include "./utils.h"

namespace dgl {

using runtime::NDArray;

namespace aten {
namespace impl {

/*!
 * \brief Check whether each row is sorted.
 */
template <typename IdType>
__global__ void _SegmentIsSorted(
    const IdType* indptr, const IdType* indices,
    int64_t num_rows, int8_t* flags) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;
  while (tx < num_rows) {
    bool f = true;
    for (IdType i = indptr[tx] + 1; f && i < indptr[tx + 1]; ++i) {
      f = (indices[i - 1] <= indices[i]);
    }
    flags[tx] = static_cast<int8_t>(f);
    tx += stride_x;
  }
}

template <DLDeviceType XPU, typename IdType>
bool CSRIsSorted(CSRMatrix csr) {
  const auto& ctx = csr.indptr->ctx;
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  auto device = runtime::DeviceAPI::Get(ctx);
  // We allocate a workspace of num_rows bytes. It wastes a little bit memory but should
  // be fine.
  int8_t* flags = static_cast<int8_t*>(device->AllocWorkspace(ctx, csr.num_rows));
  const int nt = cuda::FindNumThreads(csr.num_rows);
  const int nb = (csr.num_rows + nt - 1) / nt;
  _SegmentIsSorted<<<nb, nt, 0, thr_entry->stream>>>(
      csr.indptr.Ptr<IdType>(), csr.indices.Ptr<IdType>(),
      csr.num_rows, flags);
  bool ret = cuda::AllTrue(flags, csr.num_rows, ctx);
  device->FreeWorkspace(ctx, flags);
  return ret;
}

template bool CSRIsSorted<kDLGPU, int32_t>(CSRMatrix csr);
template bool CSRIsSorted<kDLGPU, int64_t>(CSRMatrix csr);

template <DLDeviceType XPU, typename IdType>
void CSRSort_(CSRMatrix* csr) {
  CHECK(sizeof(IdType) == 4) << "CUDA CSRSort_ does not support int64.";
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  auto device = runtime::DeviceAPI::Get(csr->indptr->ctx);
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }
  CUSPARSE_CALL(hipsparseSetStream(thr_entry->cusparse_handle, thr_entry->stream));

  NDArray indptr = csr->indptr;
  NDArray indices = csr->indices;
  const auto& ctx = indptr->ctx;
  const int64_t nnz = indices->shape[0];
  if (!aten::CSRHasData(*csr))
    csr->data = aten::Range(0, nnz, indices->dtype.bits, ctx);
  NDArray data = csr->data;

  size_t workspace_size = 0;
  CUSPARSE_CALL(hipsparseXcsrsort_bufferSizeExt(
      thr_entry->cusparse_handle,
      csr->num_rows, csr->num_cols, nnz,
      indptr.Ptr<int32_t>(), indices.Ptr<int32_t>(),
      &workspace_size));
  void* workspace = device->AllocWorkspace(ctx, workspace_size);

  hipsparseMatDescr_t descr;
  CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
  CUSPARSE_CALL(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  CUSPARSE_CALL(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));
  CUSPARSE_CALL(hipsparseXcsrsort(
      thr_entry->cusparse_handle,
      csr->num_rows, csr->num_cols, nnz,
      descr,
      indptr.Ptr<int32_t>(), indices.Ptr<int32_t>(),
      data.Ptr<int32_t>(),
      workspace));

  csr->sorted = true;

  // free resources
  CUSPARSE_CALL(hipsparseDestroyMatDescr(descr));
  device->FreeWorkspace(ctx, workspace);
}

template void CSRSort_<kDLGPU, int32_t>(CSRMatrix* csr);
template void CSRSort_<kDLGPU, int64_t>(CSRMatrix* csr);

}  // namespace impl
}  // namespace aten
}  // namespace dgl
